#include <iostream>
#include <hip/hip_runtime.h>

// CUDA plus core function
// __global__ key word indicate this function running on GPU
// threadIdx, blockIdx and blockDim is CUDA build-in in core function. used for parallel computing index

/** CUDA允许多个块执行多个线程。多个block构成一个grid */
__global__ void add(int* a, int *b, int *c, int n) {
    /** index是每个线程在整个网格中的全局索引
     * blockDim.x: x方向上一个block有多少个threads
     * blockIdx.x: 当前线程属于哪个块
     * threadIdx.x: 当前线程在所属块中的索引。
     * blockDim.x：表示每个块中的线程数。因此，blockIdx.x * blockDim.x 就表示当前块的第一个线程的全局索引。
     * 例如，如果每个块有 256 个线程，那么对于块 0（blockIdx.x = 0），
     * 第一个线程的全局索引应该是 0，对于块 1（blockIdx.x = 1），第一个线程的全局索引应该是 256。
     * threadIdx.x：表示当前线程在块内的索引。它是一个从 0 到 blockDim.x - 1 的值。因此，threadIdx.x 
     * 就是在当前块内线程的编号。
     * 
     * 对于第一个块（blockIdx.x = 0）：

线程 0 的全局索引：threadIdx.x + blockIdx.x * blockDim.x = 0 + 0 * 256 = 0
线程 1 的全局索引：threadIdx.x + blockIdx.x * blockDim.x = 1 + 0 * 256 = 1
线程 255 的全局索引：threadIdx.x + blockIdx.x * blockDim.x = 255 + 0 * 256 = 255
     */
    int index = threadIdx.x + blockIdx.x * blockDim.x; // which block in which thread.
    
    if (index < n) {
        c[index] = a[index] + b[index]; // obtain val at this position.
    }
}

int main() {
    const int arraySize = 5;
    int a[arraySize] = {1,2,3,4,5};
    int b[arraySize] = {10,20,30,40,50};
    int c[arraySize] = {0};

    // GPU memory pointer
    int *d_a, *d_b, *d_c;

    // allocate GPU memory。 void** 指向指针的指针。因为&d_a 取到的是指针的地址
    hipMalloc((void**)&d_a, arraySize * sizeof(int));
    hipMalloc((void**)&d_b, arraySize * sizeof(int));
    hipMalloc((void**)&d_c, arraySize * sizeof(int));

    // move data from CPU to GPU
    /**
     * d_a 是一个指向设备内存的指针，类型为 int*。
在调用 cudaMalloc() 时，你需要将指针的地址（&d_a）传递给该函数，这样 cudaMalloc() 可以修改这个指针，
让它指向分配到的设备内存。
由于 cudaMalloc() 需要的是 void** 类型的指针，而 d_a 是 int* 类型的指针，因此需要使用 (void**) 来进行类型转换。
     */
    hipError_t err = hipMemcpy(d_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(err) << std::endl;
    }
    hipError_t err2 = hipMemcpy(d_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (err2 != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(err2) << std::endl;
    }



    // call CUDA core function, every thread treat one ele
    int threadsPerBlock = 256; // 每个block有256个threads
    
    /** threadsPerBlock - 1 是为了向上取整。
     * 这一行代码的作用是计算 CUDA 网格（grid）中所需的块（blocks）数量。为了并行处理数据，我们通常将工作分配给多个线程，
     * 而线程被组织在块中。每个块中有多个线程，因此我们需要根据总的数据量和每个块中的线程数来确定需要多少个块。
     */
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, arraySize);

    // copy GPU back to CPU
    hipMemcpy(c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // output result
    std::cout << "Results: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // release GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
/**
 * nvcc --compiler-bindir /usr/bin/g++ plus.cu 
 * 
 * recommned to use .cu instead of .cpp, so nvcc knows how to deal with it.
 */